#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "main.h"

#define SIZE 20

int main(void) {
	int *da, *db, *dc, *ha, *hb, *hc;

	hipMalloc(&da, SIZE * sizeof(int));
	hipMalloc(&db, SIZE * sizeof(int));
	hipMalloc(&dc, SIZE * sizeof(int));
	ha = (int*)malloc(SIZE * sizeof(int));
	hb = (int*)malloc(SIZE * sizeof(int));
	hc = (int*)malloc(SIZE * sizeof(int));

	for (int i = 0; i < SIZE; i ++) {
		ha[i] = i;
		hb[i] = i;
		hc[i] = 0;
	}

	hipMemcpy(da, ha, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(db, hb, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(hc, dc, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(da);
	hipFree(db);
	hipFree(dc);
	free(ha);
	free(hb);
	free(hc);

	Timer timer;
	timer.start(L"Reading from file ...");
	unsigned int numlines = 0;
	Textfile wordlist(L"input.txt");
	unsigned int numberstrings = wordlist.countlines();
	timer.stop();

	timer.start(L"Sorting with Burstsort");
	Burstsort<Minuscule> bs;
	for (unsigned int i = 0; i < numberstrings; i++)
		bs.insert((wchar_t*)wordlist.getline(i));
	bs.sort();
	timer.stop();
	bs.print(true, false);
	bs.clear();

	wordlist.close();

	return 0;
}